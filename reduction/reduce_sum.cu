#include "hip/hip_runtime.h"
#include "error.cuh"
#include "macro.h"

#include <cassert>
#include <chrono>
#include <cmath>
#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <type_traits>

#define float float
#define WARMUP_ITER 3
#define TEST_ITER 100

using namespace std;

void reduce_sum_kernel_cpu(const float *a, float *c, size_t len) {
  double temp = 0;
  for (int i = 0; i < len; ++i) {
    temp += a[i];
  }
  *c = static_cast<float>(temp);
}

__global__ void reduce_sum_kernel_gpu_1(const float *a, float *b, size_t len) {
  __shared__ float s_tmp[THREADS_PER_BLOCK];
  auto tid = threadIdx.x;
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  double temp = 0;
  for (int i = idx; i < len; i += blockDim.x * gridDim.x) {
    temp += a[i];
  }
  s_tmp[tid] = temp;
  __syncthreads();

  int i = 2, j = 1;
  while (i <= THREADS_PER_BLOCK) {
    // tree reduction
    if ((tid % i) == 0) {
      s_tmp[tid] += s_tmp[tid + j];
    }
    __syncthreads();
    i *= 2;
    j *= 2;
  }
  if (tid == 0) {
    b[blockIdx.x] = s_tmp[0];
  }
}

__global__ void reduce_sum_kernel_gpu_2(const float *a, float *b, size_t len) {
  __shared__ float s_tmp[THREADS_PER_BLOCK];
  auto tid = threadIdx.x;
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  double temp = 0;
  for (int i = idx; i < len; i += blockDim.x * gridDim.x) {
    temp += a[i];
  }
  s_tmp[tid] = temp;
  __syncthreads();

  int offset = THREADS_PER_BLOCK / 2;
  for (; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_tmp[tid] += s_tmp[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0) {
    b[blockIdx.x] = s_tmp[0];
  }
}

__global__ void reduce_sum_kernel_gpu_3(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  __shared__ float s_tmp[THREADS_PER_BLOCK];
  auto tid = threadIdx.x;
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  double temp = 0;
  for (int i = idx; i < len; i += blockDim.x * gridDim.x) {
    temp += a[i];
  }
  s_tmp[tid] = temp;
  __syncthreads();

  int offset = THREADS_PER_BLOCK / 2;
  for (; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_tmp[tid] += s_tmp[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0) {
    atomicAdd(b, s_tmp[0]);
  }
}

__global__ void reduce_sum_kernel_gpu_4(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  __shared__ float s_tmp[THREADS_PER_BLOCK];
  auto tid = threadIdx.x;
  auto idx = (blockDim.x * 2) * blockIdx.x + threadIdx.x;
  double temp = 0;
  for (int i = idx; i < len; i += blockDim.x * 2 * gridDim.x) {
    temp += a[i] + a[i + blockDim.x];
  }
  s_tmp[tid] = temp;
  __syncthreads();

  int offset = THREADS_PER_BLOCK / 2;
  for (; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_tmp[tid] += s_tmp[tid + offset];
    }
    __syncthreads();
  }
  if (tid == 0) {
    atomicAdd(b, s_tmp[0]);
  }
}

__device__ void warp_reduce(volatile float *s_data, size_t tid) {
  s_data[tid] += s_data[tid + 32];
  s_data[tid] += s_data[tid + 16];
  s_data[tid] += s_data[tid + 8];
  s_data[tid] += s_data[tid + 4];
  s_data[tid] += s_data[tid + 2];
  s_data[tid] += s_data[tid + 1];
}

__global__ void reduce_sum_kernel_gpu_5(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  __shared__ float s_tmp[THREADS_PER_BLOCK];
  auto tid = threadIdx.x;
  auto idx = (blockDim.x * 2) * blockIdx.x + threadIdx.x;
  double temp = 0;
  for (int i = idx; i < len; i += blockDim.x * 2 * gridDim.x) {
    temp += a[i] + a[i + blockDim.x];
  }
  s_tmp[tid] = temp;
  __syncthreads();

  int offset = THREADS_PER_BLOCK / 2;
  for (; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_tmp[tid] += s_tmp[tid + offset];
    }
    __syncthreads();
  }

  if (tid < 32) {
    warp_reduce(s_tmp, tid);
  }

  if (tid == 0) {
    atomicAdd(b, s_tmp[0]);
  }
}

__global__ void reduce_sum_kernel_gpu_5_1(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  __shared__ float s_tmp[THREADS_PER_BLOCK];
  auto tid = threadIdx.x;
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  double temp = 0;
  float4 val;
  for (int i = idx; i < len / 4; i += blockDim.x * gridDim.x) {
    val = CONST_FLOAT4(a)[i];
    temp += ((val.x + val.y) + (val.z + val.w));
  }
  int i = idx + len / 4 * 4;
  if (i < len) {
    temp += a[i];
  }

  s_tmp[tid] = temp;
  __syncthreads();

  int offset = THREADS_PER_BLOCK / 2;
  for (; offset > 32; offset >>= 1) {
    if (tid < offset) {
      s_tmp[tid] += s_tmp[tid + offset];
    }
    __syncthreads();
  }

  if (tid < 32) {
    warp_reduce(s_tmp, tid);
  }

  if (tid == 0) {
    atomicAdd(b, s_tmp[0]);
  }
}

__inline__ __device__ float warp_reduce_sum_kernel(float val) {
#pragma unroll
  for (int offset = warpSize / 2; offset > 0; offset = offset >> 1) {
    val += __shfl_down_sync(FULL_MASK, val, offset);
  }

  return val;
}

__inline__ __device__ float block_reduce_sum_kernel(float val) {
  __shared__ float s_tmp[32];

  int wid = threadIdx.x / warpSize;
  int lane = threadIdx.x % warpSize;

  val = warp_reduce_sum_kernel(val);

  if (lane == 0) {
    s_tmp[wid] = val;
  }

  __syncthreads();

  val = (threadIdx.x < blockDim.x / warpSize) ? s_tmp[lane] : 0;

  if (wid == 0) {
    val = warp_reduce_sum_kernel(val);
  }

  return val;
}

__global__ void reduce_sum_kernel_gpu_6(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  auto tid = threadIdx.x;
  auto idx = (blockDim.x * 2) * blockIdx.x + threadIdx.x;
  double sum = 0;
  for (int i = idx; i < len; i += blockDim.x * 2 * gridDim.x) {
    sum += a[i] + a[i + blockDim.x];
  }

  sum = block_reduce_sum_kernel(sum);

  if (tid == 0) {
    atomicAdd(b, sum);
  }
}

__global__ void reduce_sum_kernel_gpu_6_1(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  auto tid = threadIdx.x;
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  double sum = 0;
  float4 val;
  for (int i = idx; i < len / 4; i += blockDim.x * gridDim.x) {
    val = CONST_FLOAT4(a)[i];
    sum += ((val.x + val.y) + (val.z + val.w));
  }

  sum = block_reduce_sum_kernel(sum);

  if (tid == 0) {
    atomicAdd(b, sum);
  }
}

__inline__ __device__ float warp_allreduce_sum_kernel(float val) {
#pragma unroll
  for (int offset = warpSize / 2; offset > 0; offset = offset >> 1) {
    val += __shfl_xor_sync(FULL_MASK, val, offset);
  }

  return val;
}

__inline__ __device__ float block_allreduce_sum_kernel(float val) {
  __shared__ float s_tmp[32];

  int wid = threadIdx.x / warpSize;
  int lane = threadIdx.x % warpSize;

  val = warp_allreduce_sum_kernel(val);

  if (lane == 0) {
    s_tmp[wid] = val;
  }

  __syncthreads();

  val = (threadIdx.x < blockDim.x / warpSize) ? s_tmp[lane] : 0;

  if (wid == 0) {
    val = warp_allreduce_sum_kernel(val);
  }

  return val;
}

__global__ void reduce_sum_kernel_gpu_7(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  auto tid = threadIdx.x;
  auto idx = (blockDim.x * 2) * blockIdx.x + threadIdx.x;
  double sum = 0;
  for (int i = idx; i < len; i += blockDim.x * 2 * gridDim.x) {
    sum += a[i] + a[i + blockDim.x];
  }

  sum = block_allreduce_sum_kernel(sum);

  if (tid == 0) {
    atomicAdd(b, sum);
  }
}

__global__ void reduce_sum_kernel_gpu_7_1(const float *a, float *b, size_t len) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    b[0] = 0;
  }
  auto tid = threadIdx.x;
  auto idx = blockDim.x * blockIdx.x + threadIdx.x;
  double sum = 0;
  float4 val;
  for (int i = idx; i < len / 4; i += blockDim.x * gridDim.x) {
    val = CONST_FLOAT4(a)[i];
    sum += ((val.x + val.y) + (val.z + val.w));
  }

  sum = block_allreduce_sum_kernel(sum);

  if (tid == 0) {
    atomicAdd(b, sum);
  }
}

void init_random(float *data, size_t len) {
  for (size_t i = 0; i < len; ++i) {
    data[i] =
        static_cast<float>(std::rand()) / static_cast<float>(RAND_MAX);
  }
}

void compare(float res1, float res2) {
  double diff = std::abs(res1 - res2);
  if (std::is_same<float, double>::value) {
    if (diff > 1e-7) {
      fprintf(stderr, "check failed ! res1: %.13f res2: %.13f\n", res1, res2);
      return;
    }
  } else {
    if (diff > 1e-4) {
      fprintf(stderr, "check failed ! res1: %.7f res2: %.7f\n", res1, res2);
      return;
    }
  }
  printf("check pass !\n");
}

void reduce_sum_gpu_1(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_1<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_b, len);
  CHECK_CUDA_ERROR(hipGetLastError());
  reduce_sum_kernel_gpu_1<<<1, THREADS_PER_BLOCK>>>(d_b, d_c, THREADS_PER_BLOCK);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_2(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_2<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_b, len);
  CHECK_CUDA_ERROR(hipGetLastError());
  reduce_sum_kernel_gpu_2<<<1, THREADS_PER_BLOCK>>>(d_b, d_c, THREADS_PER_BLOCK);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_3(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_3<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_4(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_4<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_5(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_5<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_5_1(const float *d_a, float *d_b, float *d_c,
                    size_t len) {
  reduce_sum_kernel_gpu_5_1<<<
      MIN((len / 4 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 2048),
      THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_6(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_6<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_6_1(const float *d_a, float *d_b, float *d_c,
                    size_t len) {
  reduce_sum_kernel_gpu_6_1<<<
      MIN((len / 4 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 2048),
      THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_7(const float *d_a, float *d_b, float *d_c,
                  size_t len) {
  reduce_sum_kernel_gpu_7<<<BLOCKS_PER_GRID(len), THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

void reduce_sum_gpu_7_1(const float *d_a, float *d_b, float *d_c,
                    size_t len) {
  reduce_sum_kernel_gpu_7_1<<<
      MIN((len / 4 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 2048),
      THREADS_PER_BLOCK>>>(d_a, d_c, len);
  CHECK_CUDA_ERROR(hipGetLastError());
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    std::cout << "[Useage] " << argv[0] << " "
              << "<data len>" << std::endl;
    std::exit(1);
  }
  std::srand(std::time(nullptr));

  size_t len = atol(argv[1]);
  size_t size = len * sizeof(float);

  float *a = (float *)malloc(size);
  float *cpu_c = (float *)malloc(1 * sizeof(float));
  float *gpu_c = (float *)malloc(1 * sizeof(float));

  init_random(a, len);

  float *d_a, *d_b, *d_c;
  CHECK_CUDA_ERROR(hipMalloc(&d_a, size));
  CHECK_CUDA_ERROR(hipMalloc(&d_b, sizeof(float) * THREADS_PER_BLOCK));
  CHECK_CUDA_ERROR(hipMalloc(&d_c, sizeof(float) * 1));

  CHECK_CUDA_ERROR(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));

  for (int i = 0; i < WARMUP_ITER; ++i) {
    reduce_sum_gpu_1(d_a, d_b, d_c, len);
  }

  reduce_sum_kernel_cpu(a, cpu_c, len);

  BENCHMARK(reduce_sum_gpu_1, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_2, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_3, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_4, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_5, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_5_1, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_6, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_6_1, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_7, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  BENCHMARK(reduce_sum_gpu_7_1, TEST_ITER, d_a, d_b, d_c, len);
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_c, d_c, sizeof(float) * 1, hipMemcpyDeviceToHost));
  compare(*gpu_c, *cpu_c);

  CHECK_CUDA_ERROR(hipFree(d_a));
  CHECK_CUDA_ERROR(hipFree(d_b));
  CHECK_CUDA_ERROR(hipFree(d_c));

  return 0;
}